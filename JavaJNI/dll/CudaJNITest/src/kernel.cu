#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <malloc.h> 
#include <stdio.h>
#include "CudaJNI.h"
static int k = 0;
__global__ void helloFromGPU() {
	printf("Hello World from GPU! this is block (%d,%d,%d) thread (%d,%d,%d)\n",
		blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void printMem(int* p) {
	printf("Hello World from GPU!");
	for (int i = 0; i < 10; i++) {
		printf("this is %d\n", p[i]);
	}
}
__global__ void printID() {
	printf("Hello World from GPU! this is thread %d ");
}
int printHelloWorld(void) {

	int* d_data = 0;
	int h_data[] = { 1,2,3,4,5,6,7,8,9,10 };
	hipError_t cudaStatus = hipMalloc((void**)& d_data, 10 * sizeof(int));
	cudaStatus = hipMemset(&d_data, 3, 10 * sizeof(int));
	helloFromGPU << <2, 5 >> > ();
	hipDeviceReset();
	char z;
	scanf("%c", &z);
Error:
	return 0;
	return 0;
}
JNIEXPORT jint JNICALL Java_CudaJNI_printHelloWorldByGPU
(JNIEnv*, jobject) {
	printHelloWorld();
	return 1;
}