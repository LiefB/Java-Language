#include "hip/hip_runtime.h"
#include "calculate.cuh"

__global__ void gpuCalculate(int* d_arr) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    d_arr[idx]++;
}

void callCudaFunc(int* carr, int n) {
    int* d_arr;
    hipMalloc((void**)& d_arr, n * sizeof(int));
    hipMemcpy(d_arr, carr, n*sizeof(int), hipMemcpyHostToDevice);

    gpuCalculate<<<100000, 1000>>>(d_arr);

    hipMemcpy(carr, d_arr, n*sizeof(int), hipMemcpyDeviceToHost);
}

