#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctime>
#include "calculate.cuh"

/*
__global__ void gpuCalculate(int* d_arr) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    d_arr[idx]++;
}

void callCudaFunc(int* carr, int n) {
    int* d_arr;
    hipMalloc((void**)& d_arr, n * sizeof(int));
    hipMemcpy(d_arr, carr, n*sizeof(int), hipMemcpyHostToDevice);

    gpuCalculate<<<100000, 1000>>>(d_arr);

    hipMemcpy(carr, d_arr, n*sizeof(int), hipMemcpyHostToDevice);
}
*/

int main(int argc, char** argv) {
    int n = 100000000;
    int* arr = new int[100000000];
    for (int i = 0; i < n; i++) {
        arr[i] = i;
    }

    clock_t startTime = clock();
    callCudaFunc(arr, n);
    clock_t endTime = clock();
    printf("%lf\n", (double)(endTime - startTime)/CLOCKS_PER_SEC);

    for (int i = 0; i < 10; i++) {
        printf("%d\n", arr[i]);
    }    

    for (int i = 0; i < n; i++) {
        if (arr[i] != i + 1) {
	    printf("Something Wrong......\n");
            break;
        }
    }
    return 0;
}
