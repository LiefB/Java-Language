#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>
#include <jni.h>
#include "JNI_Func.h"

// 考虑安全性，不直接暴露Java堆上内存地址，而是返回一个数组副本，再将副本复制回Java堆中真实数组中
/*
JNIEXPORT void JNICALL Java_JNI_1Func_calculate
  (JNIEnv * env, jobject obj, jintArray arr, jint n) {
        jint* carr;
        carr = env->GetIntArrayElements(arr, 0);
        if (carr == NULL)
                return;
        for (int i = 0; i < n; i++) {
                carr[i]++;
        }
        env->ReleaseIntArrayElements(arr, carr, 0);
}
*/


__global__ void gpuCalculate(int* d_arr) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	//if (idx < 100000000)
	d_arr[idx]++;
}

void callCudaFunc(int* carr, int n) {
	int* d_arr;
	hipMalloc((void**)& d_arr, n * sizeof(int));
	hipMemcpy(d_arr, carr, n*sizeof(int), hipMemcpyHostToDevice);

	gpuCalculate <<<100000, 1000 >>> (d_arr);

	hipMemcpy(carr, d_arr, n*sizeof(int), hipMemcpyDeviceToHost);
}

// 创建一个阻止GC的临界区，将数组的真实数据直接暴露给用户
JNIEXPORT void JNICALL Java_JNI_1Func_calculate
  (JNIEnv * env, jobject obj, jintArray arr, jint n) {
        int* carr = static_cast<int*>(env->GetPrimitiveArrayCritical(arr, 0));

        callCudaFunc(carr, n);

        env->ReleasePrimitiveArrayCritical(arr, carr, 0);
}
